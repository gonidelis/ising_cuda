
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <math.h>

#define THREADS_PER_BLOCK 50


void validation(int n,int k,int *expected,int *G){
  int flag=0;
	for(int v = 0; v < n*n; v++){
    if(expected[v] != G[v]){
      flag=-1;
    }
  }
  if(flag==0){
    printf("k=%d: CORRECT ISING MODEL\n",k);
  }else{
    printf("k=%d: WRONG ISING MODEL\n",k);
  }
}



__global__ void calc_moment(int num_of_moments,int num_of_blocks, int n,int *G,int *newG,double *w){

  int x,y; //indices of a moment
  double infl; //temporary value to define the influence of the neighbors and the new value of each moment

  //Find the id of the current thread
  int id=blockIdx.x*blockDim.x+threadIdx.x;

  //Current moment calculated from a thread
  //int current=id+THREADS_PER_BLOCK*num_of_blocks;
  // Calculate thread_id based on the coordinates of the block
  int blockX = blockIdx.x % num_of_blocks;
  int blockY = blockIdx.x / num_of_blocks;
  int thread_id = blockX * THREADS_PER_BLOCK + blockY * n * THREADS_PER_BLOCK + threadIdx.x;

  int max=THREADS_PER_BLOCK*THREADS_PER_BLOCK*num_of_moments*num_of_blocks;
  int step=THREADS_PER_BLOCK*num_of_blocks;

  if(thread_id<n*n){

    for(int moment=thread_id; moment<thread_id + n * THREADS_PER_BLOCK && moment<n*n; moment+=n){
      infl=0;
      //Find coordinates x,y of each moment
      //i -> x coordinate
      //j -> y coordinate
      int i=moment%n;
      int j=moment/n;


      //for all the neighbors
      for(int c=0;c<5;c++){
        for(int d=0;d<5;d++){

          //Do not update if the next neighbor coincides with the current point
          if((c!=2) || (d!=2)){

            //Windows centered on the edge lattice points wrap around to the other side
            y = ((c-2)+i+n) % n;
            x = ((d-2)+j+n) % n;

            //Influence of a neighbor is increased
            //Add to infl the weight*value of the previous neighbor
            infl += G[y*n+x] * w[c*5+d];

          }
        }
      }

      //Next value of a moment is defined according to the value of infl
      if(infl>0.0001){
        newG[i*n+j]=1;
      }else if(infl<-0.0001){
        newG[i*n+j]=-1;
      }else{
        newG[i*n+j]=G[i*n+j];
      }
    }
  }
}



void ising( int *G, double *w, int k, int n){

  int *newG,*swapG;
  hipMallocManaged(&newG,n*n*sizeof(int)); //save previous G before changing it

  int num_of_blocks;
  if(n%THREADS_PER_BLOCK==0){
    num_of_blocks=n/THREADS_PER_BLOCK;
  }
  else{
    num_of_blocks=n/THREADS_PER_BLOCK+1;
  }

  //if n*n % num_of_blocks * THREADS_PER_BLOCK !=0 take the ceiling value
  int num_of_moments = (n*n)/(num_of_blocks *num_of_blocks * THREADS_PER_BLOCK);

  //for every iteration (k)
  for(int t=0;t<k;t++){

    //Call 11^2 bllocks with 47 threads per block
    //Each thread calculates 11 moments
    calc_moment<<<num_of_blocks*num_of_blocks,THREADS_PER_BLOCK>>>(num_of_moments,num_of_blocks,n,G,newG,w);

    // Synchronize threads before swapping the arrays
		hipDeviceSynchronize();

    //Swap arrays G and newG
    swapG=newG;
    newG=G;
    G=swapG;

  }

  //If last k is an odd number, then the returned G should be newG
  if(k % 2 == 1){
    memcpy(newG, G, n*n*sizeof(int));
  }

  //cudaFree(newG);
}




int main(){

	// n = dimentions  k = number of iterations
	int n = 517;	int k = 1;

  struct timeval start, end;
  gettimeofday(&start,NULL);

  // Array of weights
  double *weights;
  hipMallocManaged(&weights,5*5*sizeof(double));
  double w[25] = {0.004, 0.016, 0.026, 0.016, 0.004,
                  0.016, 0.071, 0.117, 0.071, 0.016,
                  0.026, 0.117, 0, 0.117, 0.026,
                  0.016, 0.071, 0.117, 0.071, 0.016,
                  0.004, 0.016, 0.026, 0.016, 0.004};
  memcpy(weights,w,sizeof(w));


	// Get the moments of array G from the binary file
  FILE *fptr = fopen("conf-init.bin","rb");
  if (fptr == NULL){
      printf("Error: Cannnot open file");
      exit(1);
  }
  int *G;
  hipMallocManaged(&G,n*n*sizeof(int));
  fread(G, sizeof(int), n*n, fptr);
  fclose(fptr);


  //Save a copy of G to call again function ising() for different k
  //because ising() is changing the array G
  int *copyG;
  hipMallocManaged(&copyG,n*n*sizeof(int));
  memcpy(copyG, G, n*n*sizeof(int));


  //Call ising for k=1
  ising(G, weights, k, n);
	// Check results by comparing with ready data for k=1
	int *expected;
  hipMallocManaged(&expected,n*n*sizeof(int));
	fptr = fopen("conf-1.bin","rb");
  if (fptr == NULL){
      printf("Error: Cannnot open file");
      exit(1);
  }
	fread(expected, sizeof(int), n*n, fptr);
	fclose(fptr);
  validation(n,k,expected,G);


  //Call ising for k=4
  k=4;
  memcpy(G, copyG, n*n*sizeof(int));
  ising(G, weights, k, n);
	// Check for k = 4
	fptr = fopen("conf-4.bin","rb");
  if (fptr == NULL){
      printf("Error: Cannnot open file");
      exit(1);
  }
	fread(expected, sizeof(int), n*n, fptr);
	fclose(fptr);
	validation(n,k,expected,G);


  //Call ising for k=11;
  k=11;
  memcpy(G, copyG, n*n*sizeof(int));
  ising(G, weights, k, n);
	// Check for k = 11
	fptr = fopen("conf-11.bin","rb");
  if (fptr == NULL){
      printf("Error: Cannnot open file");
      exit(1);
  }
	fread(expected, sizeof(int), n*n, fptr);
	fclose(fptr);
	validation(n,k,expected,G);

  //cudaFree(G);
  //cudaFree(copyG);
  //cudaFree(expected);

  gettimeofday(&end,NULL);
  printf("time : %lf\n", (double)((end.tv_usec - start.tv_usec)/1.0e6 + end.tv_sec - start.tv_sec));


  return 0;
}
